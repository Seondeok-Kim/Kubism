#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "common.h"
#include "cpu_function.h"
#include "gpu_function.cuh"

int main() {
    int num_clusters = 18;
    int num_data_points = 6;

    // Create distance array
    float* h_distance;
    hipHostAlloc((void**)&h_distance, num_clusters * num_data_points * sizeof(float), hipHostMallocMapped);
    memset(h_distance, 0, num_clusters * num_data_points * sizeof(float));

    float* d_distance;
    hipHostGetDevicePointer(&d_distance, h_distance, 0);

    // initialize skip array (store the centroids to be skipped)
    int* h_skip;
    hipHostAlloc((void**)&h_skip, num_clusters * num_data_points * sizeof(int), hipHostMallocMapped);
    init_skip(h_skip, num_clusters, num_data_points);
    printf("\n=== Clusters to calculate ===\n");
    print_clusters_to_calculate(h_skip, num_clusters, num_data_points);

    int* d_skip;
    hipHostGetDevicePointer(&d_skip, h_skip, 0);

    // Creat bitmask array
    int bitmask_col = (num_clusters + BITMASK_SIZE - 1) / BITMASK_SIZE;
    BITMASK* h_bitmask;
    hipHostAlloc((void**)&h_bitmask, num_data_points * bitmask_col * sizeof(BITMASK), hipHostMallocMapped);

    BITMASK* d_bitmask;
    hipHostGetDevicePointer(&d_bitmask, h_bitmask, 0);

    // Marking
    marking<<<1, 512>>>(d_bitmask, d_skip, num_clusters, num_data_points);
    hipDeviceSynchronize();
    printf("\n=== Bitmask (LSB to MSB) ===\n");
    print_bitmask(h_bitmask, num_clusters, num_data_points);

    // Distance calculation
    printf("\n=== Distance Calculation Start ===\n");
    calculate_distance<<<1, 512>>>(d_distance, d_bitmask, num_clusters, num_data_points);
    hipDeviceSynchronize();


    // Distance calculation results
    printf("\n=== Distance Calculation Results ===\n");
    print_distance(h_distance, num_clusters, num_data_points);

    hipHostFree(d_bitmask);
    hipHostFree(d_skip);

    return 0;
}
